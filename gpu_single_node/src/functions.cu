#include "hip/hip_runtime.h"
#include <thrust/advance.h>
#include <thrust/system_error.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/binary_search.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <math.h> //for math
//#include <gmpxx.h> //for precision calculation
#include <vector> //to hold search results
#include <stdio.h>
#include <fstream>
#include <iterator>
#include <iomanip>
#include <algorithm> //compute max of vector
#include <numeric> //compute sum of vector (accumulate)
#include <omp.h>
#include "structures.h"
//speed not important for final statistics, so optimising this is silly
#define PI 3.14159265359
//this file contains the main statistics functions
//This file relies on main's importing of the logFacts data file

//this number is the max number of threads in a block on
//guillimin
#define BLOCK_SIZE = 1024

//extern tells the compiler that this variable is global and
//is already initialized elsewhere
extern double *logFacts;
extern int maxFact;

using namespace std;

//Forward declaration for use in class
double log_odds_ratio(double *counts, int length, int m_max, 
					  double nu, double nudot, bool verbose);


//This function returns the choose function
double log_choose(int first, int second)
{
	//sanity check
	if (first > maxFact || second > maxFact || second > first)
	{
		return 0;
	}
	//the log of the choose function
	return logFacts[first] - logFacts[second] - logFacts[first-second];
}

//this function normalizes a list of counts to start at 0 and 
//be in terms of seconds
void normalize_counts(double *counts, int length)
{
	double t_min = counts[0];
	for (int i = 0; i < length; i ++)
	{
		counts[i] -= t_min;
	}
}

//This function gets the number of events which occur within
//the specified range. Length is the total number of events
int num_events(double *counts, int length, double start, double end)
{
	//the number of events in this region of the counts
	int num = 0;
	//go through all counts
	//note that this assumes they are in order
	for (int i = 0; i < length; i ++)
	{
		//check if the count is in the region
		if (counts[i] >= start && counts[i] < end)
		{
			num ++;
		}
		//if the count is later, then we choose to not be redundant
		else if (counts[i] >= end)
		{
			break;
		}
	}
	//return the number of events
	return num;
}

/*
//CUDA kernel to create n_mvals*length matrix of bins
__global__ void create_binnings(double *counts, int *mvals,
								int n_mvals, double nu, double nudot,
								unsigned char **binning)
{
	//threads=length
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	double t = counts[idx];
	unsigned char tmp_bin = 0;
	for (int i = 0; i < n_mvals; i++)
	{
		tmp_bin = (unsigned char)((int)(fmod(t*(nu+0.5*t*nudot),1)*mvals[i]));
		binning[i][idx] = tmp_bin;
		binning[i][idx] = 54;

	}
	//n[(int)(fmod(counts[i]*(nu+0.5*counts[i]*nudot),1)*m)]++;
}
*/
//function gets decimal portion of double
__device__ double get_decimal (double x) {return x - (int)x;}

__global__ void create_binnings(double *counts, int *mvals,
								int length,
								int n_mvals, double nu, double nudot,
								unsigned char *binning)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < length)
	{
		double t = counts[idx];
		int index = idx;
		unsigned char tmp_bin = 0;
		for (int i = 0; i < n_mvals; i++)
		{
			tmp_bin = (int)(get_decimal(t*(nu+0.5*t*nudot))*mvals[i]);
			binning[index] = tmp_bin;
			index += length;
		}
	}
}

//function makes CUDA calls
unsigned char *get_bins(double *counts_d, int length, double *counts_h,
						  int *mvals_d, int *mvals_h, int n_mvals, double nu, double nudot)
{
	unsigned char *binning_h;
	unsigned char *binning_d;
	//initialize thrust arrays
	binning_h = new unsigned char [n_mvals*length];
	binning_h[0] = 100;
	hipError_t error;
	hipMalloc((void**)&counts_d,length*sizeof(double));		
	hipMalloc((void**)&mvals_d,n_mvals*sizeof(double));
	error = hipMalloc((void**)&binning_d,n_mvals*length);
	if (error!=hipSuccess) {printf("Error! %s\n",hipGetErrorString(error));}
	printf("Copying data...\n");
	hipMemcpy(counts_d,counts_h,length*sizeof(double),
	 		   hipMemcpyHostToDevice);
	hipMemcpy(mvals_d,mvals_h,n_mvals*sizeof(double),
	 		   hipMemcpyHostToDevice);
	error = hipGetLastError();
	if (error!=hipSuccess) {printf("Error! %s\n",hipGetErrorString(error));}
	else {printf("Memory Allocated!\n");}
	printf("Binning data...\n");
	create_binnings<<<40285,1024>>>(counts_d, mvals_d, length, n_mvals, nu, nudot, binning_d);
	thrust::sort(counts_d,counts_d+length);
	//error = hipDeviceSynchronize();	
	if (error!=hipSuccess) {printf("Error! %s\n",hipGetErrorString(error));}
	error = hipMemcpy(binning_h,binning_d,n_mvals*length*sizeof(unsigned char),
			   hipMemcpyDeviceToHost);
	//error = hipGetLastError();
	if (error!=hipSuccess) {printf("Error! %s\n",hipGetErrorString(error));}
	printf("Done GPU. Cleaning up...\n");
	hipFree(binning_d);
	hipFree(counts_d);
	hipFree(mvals_d);
	return binning_h;
}

/*__global__ void t_bin_counts(thrust::device_vector<double> counts,
							 thrust::device_vector<unsigned char> t_binning,
							 double nu, double nudot,
							 thrust::device_vector<int> mvals)
							 */
__global__ void t_bin_counts(double* counts, int length,
							 unsigned char* t_binning,
							 double nu, double nudot,
							 int* mvals, int n_mvals)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < length)
	{
		double t = counts[idx];
		int index = idx;
		unsigned char tmp_bin = 0;
		for (int i = 0; i < n_mvals; i++)
		{
			tmp_bin = (unsigned char)(get_decimal(t*(nu+0.5*t*nudot))*mvals[i]);
			t_binning[index] = tmp_bin;
			index += length;
		}
	}
}

__global__ void t_bin_counts_two(double* counts, int length,
							 unsigned char* t_binning,
							 double nu, double nudot)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < length)
	{
		t_binning[idx] = (unsigned char)(get_decimal(counts[idx]*(nu+0.5*counts[idx]*nudot))*256);
	}
}

__global__ void count_bins(unsigned char *bins, int *histogram_ss, int length)
{

	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < length-1)
	{
		if (bins[idx] != bins[idx+1])
		{
			//log end of one
			histogram_ss[256+bins[idx]] = idx;
			//log start of other
			histogram_ss[bins[idx+1]] = idx+1;
		}
	}
}
__global__ void get_histo (int *histogram, int *histogram_ss)
{
	int idx = threadIdx.x;
	if (histogram_ss[256+idx] != -1 && histogram_ss[idx] != -1)
		histogram[idx] = histogram_ss[256+idx] - histogram[idx];
}
//function reduces bins by a factor of two
__global__ void reduce_bins_two(int* bins)
{
	int idx = threadIdx.x;	
	bins[idx] = bins[2*idx]+bins[2*idx+1];
}

__global__ void fake_bins(unsigned char *t_binning, int length)
{
	int idx = threadIdx.x;
	t_binning[length+idx] = (unsigned char) idx;
}

__device__ void reduce_bins(int* bins, int m)
{
    for (int i = 0; i < m; i ++)
    {
        bins[i] = bins[2*i]+bins[2*i+1];
    }
}

__global__ void best_five(double *counts, int length,// double *odds_d, 
                          unsigned long long per, double nu_min,
                          double d_nu, double *logFacts_d)
                         // double *nus_d)
{
    //get ID of this core.
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
    //printf("idx=%d\n",idx);
    //make sure last piece
	if (idx < length-1)
	{
        double odds, om1, nudot;
        int m, bins[256]={0};
        //double best[5][3]={0};
        double start = per*d_nu*idx+nu_min;
        double end = start + per*d_nu;
		for (double
		     nu =  start;
			 nu <= end;
			 nu += d_nu)
        {
            odds = 0;
            om1 = 0;
            nudot = 0;
            //nudot=-Pdot/P^2=-v^2*Pdot
            //d_nudot=-nu^2*d_nudot
            //dPdot=Pmin/T^2*P=1/(numax*T^2*nu)
            //for (double
            //        nudot =  nudot_min;
            //        nudot <= nudot_max;
            //        nudot += d_nudot)
            {
                for (int i = 0; i < 256; i++)
                {
                    bins[i] = 0;
                }
                for (int i = 0; i < length; i ++)
                {
                    //With nudot
                    //bins[(unsigned char)(get_decimal(counts[i]*(nu+0.5*counts[i]*nudot))*256)]++;
                    //without nudot
                    bins[(unsigned char)(get_decimal(counts[i]*nu)*256)]++;
                }
                m = 256;
                odds = 0;
                om1 = 0;
                for (int j = 0; j < 256; j++)
                {
                    om1+=logFacts_d[bins[j]];
                }
                om1  += logFacts_d[255]-logFacts_d[length+255]+((double)length)*__log(256.0);
                odds += __exp(om1);
                for (int k = 1; k < 8; k++)
                {
                    m = m >> 1;
                    //make the pointers
                    reduce_bins(bins,m);
                    //	histogram.resize(m);
                    //	binned.resize(m);
                    om1 = 0;
                    //for (int j = 0; j < m; j++)
                    //printf("%d,",binned[j]);
                    //printf("\n");
                    for (int j = 0; j < m; j++)
                    {
                        om1+=logFacts_d[bins[j]];
                    }
                    om1  += logFacts_d[m-1]-logFacts_d[length+m-1]+((double)length)*log((double)m);
                    odds += exp(om1);
                }
                //if (odds > 1e-3)
                odds /= 8;
                odds *= d_nu/nu;
                //put in new best.
				if (odds > 0.1)
				{
                    printf("nu = %e, odds= %e\n", nu, odds);
                    /*
                    //printf("OREONO\n");
					for (int i = 3; i >= 0; i --)
					{
						if (odds < best[i][0])
						{
							for (int j = 3; j >= i + 1; j--) 
							{
								best[j+1][0] = best[j][0];
								best[j+1][1] = best[j][1];
								best[j+1][2] = best[j][2];
							}
							best[i+1][0] = odds;
							best[i+1][1] = nu;
							best[i+1][2] = nudot;
							break;
						}
						else if (i == 0)
						{
							for (int j = 3; j >= 0; j--) 
							{
								best[j+1][0] = best[j][0];
								best[j+1][1] = best[j][1];
								best[j+1][2] = best[j][2];
							}
							best[0][0] = odds;
							best[0][1] = nu;
							best[0][2] = nudot;
						}
					}
                    */
				}
            }
        }
        /*
        for (int i = 0; i < 5; i++)
        {
            odds_d[i+idx*5] = best[i][0];
            nus_d[i+idx*5]  = best[i][1];
        }
        */
    }
    return;
}


double t_odds_two(double *counts_h, int length,
                  double nu_min, double nu_max,
                  double nudot_min, double nudot_max,
                  int verbosity, const char* filename)
{
    try
    {
        //GTX 970 has 1664 cuda cores. 832 per block
        int cores = 1664;
        double d_nu = 1/counts_h[length-1];
        printf("Loading counts...\n");
        thrust::device_vector<double> counts_d(counts_h, counts_h+length);
        double *counts_d_pointer = thrust::raw_pointer_cast(counts_d.data());
        printf("Loading factorials...\n");
        thrust::device_vector<double> logFacts_d(logFacts, logFacts+maxFact);
        double *logFacts_d_pointer = thrust::raw_pointer_cast(logFacts_d.data());

        //calculate number of frequencies to iterate
        unsigned long long op = (unsigned long long)(nu_max-nu_min)/(d_nu);
        unsigned long long per = (unsigned long long)(op/1664.);
        //each thread gets 5 odds allocated to it. Should give the best.
        /*
        thrust::device_vector<double> odds_d(5*cores);
        double *odds_pointer = thrust::raw_pointer_cast(odds_d.data());
        thrust::device_vector<double>   nus_d(5*cores);
        double *nus_pointer = thrust::raw_pointer_cast(nus_d.data());
        */
        printf("Starting search!\n");
        best_five<<<2,1024>>>(counts_d_pointer,length,//odds_pointer,
                per,
                nu_min,d_nu,logFacts_d_pointer);//,nus_pointer);
	    hipError_t error = hipDeviceSynchronize();	
	    if (error!=hipSuccess) {printf("Error! %s\n",hipGetErrorString(error));}
        printf("Search complete\n");
        //printf("Search complete. Retrieving results.\n");
        /*
        thrust::host_vector<double> odds_h = odds_d;
        thrust::host_vector<double>  nus_h =  nus_d;
        for (int i = 0; i < nus_h.size(); i++)
        {
            if (odds_h[i] > 0.1)
                printf("odds = %e, nu = %e\n",odds_h[i],nus_h[i]);
        }
        */
        counts_d.clear();
        counts_d.shrink_to_fit();
        //odds_d.clear();
        //odds_d.shrink_to_fit();
        logFacts_d.clear();
        logFacts_d.shrink_to_fit();
        //nus_d.clear();
        //nus_d.shrink_to_fit();
        /*
        odds_h.clear();
        odds_h.shrink_to_fit();
        nus_h.clear();
        nus_h.shrink_to_fit();
        */
    }
    catch(thrust::system_error &e)
    {
        std::cerr << "Error: "<<e.what() << std::endl;
        exit(-1);
    }
    return 0;
}
				 

double t_odds_two_old(double *counts_h, int length,
				      double nu_min, double nu_max,
				      double nudot_min, double nudot_max,
				      int verbosity, const char* filename)
{
	//the entered mvals should be 2^1 up to 2^8
	try
	{
		double d_nu = 1/counts_h[length-1];
		//double d_nu = 1e-5;
		//double d_nudot = 1e-8;
		//printf("Length: %d\n",length);
		thrust::device_vector<double> counts_d(counts_h, counts_h+length);
		thrust::device_vector<unsigned char> t_binning(length,0);
		thrust::host_vector<int> binned(256,0);
		double *counts_d_pointer = thrust::raw_pointer_cast(counts_d.data());
		unsigned char *t_binning_pointer = thrust::raw_pointer_cast(t_binning.data());
		unsigned int blocks = (unsigned int)(length/1024.0 + 1.0);
		unsigned long counteri = 0;
		double odds = 0;
		double om1 = 0;
		int m;
		int counter = 0;
		double best[5][3] = {0};
		unsigned long opct = (unsigned long)(0.01*(nu_max-nu_min)/d_nu);
		for (double
		     nu =  nu_min;
			 nu <= nu_max;
			 nu += d_nu)
		{
		//	for (double
		//		 nudot =  nudot_min;
		//		 nudot <= nudot_max;
		//		 nudot += d_nudot)
			
			//double d_pdot = d_nu*d_nu/(nu_max*nu);
			double d_nudot = nu*d_nu*d_nu/(nu_max);
			//nudot=-Pdot/P^2=-v^2*Pdot
			//d_nudot=-nu^2*d_nudot
			//dPdot=Pmin/T^2*P=1/(numax*T^2*nu)
			for (double
				 nudot =  nudot_min;
				 nudot <= nudot_max;
				 nudot += d_nudot)
			{

				counteri ++;
				if (counteri >= opct)
				{
					counteri = 0;
					printf("%f percent of the way.\n",100.0*(nu-nu_min)/(nu_max-nu_min));
					ofstream file(filename, ios::app);
					file << "range,"; 
					file << scientific << setprecision(10) << nu_min << "-";
					file << scientific << setprecision(10) << nu << "!";
					file << scientific << setprecision(10) << nu_max << ",";
					file << scientific << nudot_min << "-";
					file << scientific << nudot_max << "\n";
					for (int i = 0; i < 5; i ++)
					{
						//printf("The %dth best odds are %e for a nu of %.9e and nudot -%.9e\n",
						//i+1,best[i][0],best[i][1],best[i][2]);
						file << scientific << best[i][0] << ",";
						file << scientific << setprecision(10) << best[i][1] << ",";
						file << scientific << -best[i][2];
						file << "\n";
					}
					file.close();
				}
				t_bin_counts_two<<<blocks,1024>>>(counts_d_pointer, length, t_binning_pointer, nu, nudot);
				thrust::sort(t_binning.begin(), t_binning.end());
				thrust::device_vector<int> histogram(256,0);

				thrust::counting_iterator<int> search_begin(0);
				thrust::upper_bound(t_binning.begin(), t_binning.end(),
									search_begin, search_begin + 256,
									histogram.begin());
				thrust::adjacent_difference(histogram.begin(), histogram.end(),
										    histogram.begin());
				binned=histogram;
				m = 256;
				odds = 0;
				om1 = 0;
				for (int j = 0; j < 256; j++)
				{
					om1+=logFacts[binned[j]];
				}
				om1  += logFacts[255]-logFacts[length+255]+((double)length)*log(256);
				odds += exp(om1);
				for (int k = 1; k < 8; k++)
				{
					/*
					for (int x = 0; x < m; x ++)
					{
						printf("%d,",binned[x]);
					}
					printf("\n");
					*/
					m = m >> 1;
					//printf("m=%d\n",m);
					//make the pointers
					int *bins_d = thrust::raw_pointer_cast(histogram.data());
					reduce_bins_two<<<1,m>>>(bins_d);
				//	histogram.resize(m);
				//	binned.resize(m);
					binned = histogram;
					om1 = 0;
					//for (int j = 0; j < m; j++)
					//printf("%d,",binned[j]);
					//printf("\n");
					for (int j = 0; j < m; j++)
					{
						om1+=logFacts[binned[j]];
					}
					om1  += logFacts[m-1]-logFacts[length+m-1]+((double)length)*log(m);
					odds += exp(om1);
				}
				//if (odds > 1e-3)
				odds /= 8;
				odds *= d_nu/nu;
				//results.nu.push_back(nu);
				//results.nudot.push_back(nudot);
				//results.odds.push_back(odds);
				//if (counter %50000==0 || odds > 1e-4)
				/*
				if (verbosity == 2 || (verbosity == 1 && odds > 1e-3) || (verbosity == 0 && odds > 1e-1))
				{
					printf("Search %d gives odds of %e for nu %.9e and nudot -%.9e\n",counter,odds,nu,nudot);
				}
				else if (verbosity == 1 && counter%50000==0)
				{
					printf("On search %d, and nu=%.9e Hz\n",counter,nu);	
				}
				*/
				
				if (odds > best[4][0])
				{
					for (int i = 3; i >= 0; i --)
					{
						if (odds < best[i][0])
						{
							for (int j = 3; j >= i + 1; j--) 
							{
								best[j+1][0] = best[j][0];
								best[j+1][1] = best[j][1];
								best[j+1][2] = best[j][2];
							}
							best[i+1][0] = odds;
							best[i+1][1] = nu;
							best[i+1][2] = nudot;
							break;
						}
						else if (i == 0)
						{
							for (int j = 3; j >= 0; j--) 
							{
								best[j+1][0] = best[j][0];
								best[j+1][1] = best[j][1];
								best[j+1][2] = best[j][2];
							}
							best[0][0] = odds;
							best[0][1] = nu;
							best[0][2] = nudot;
						}
					}
				}
			}
		}
		//clear up space
		counts_d.clear();
		counts_d.shrink_to_fit();
		ofstream file(filename, ios::app);
		file << "range,"; 
		file << scientific << setprecision(10) << nu_min << "-";
		file << scientific << setprecision(10) << nu_max << ",";
		file << scientific << nudot_min << "-";
		file << scientific << nudot_max << "\n";
		for (int i = 0; i < 5; i ++)
		{
			//printf("The %dth best odds are %e for a nu of %.9e and nudot -%.9e\n",
				   //i+1,best[i][0],best[i][1],best[i][2]);
			file << scientific << best[i][0] << ",";
			file << scientific << setprecision(10) << best[i][1] << ",";
			file << scientific << -best[i][2];
			file << "\n";
		}
		file.close();
		//results.write_best(10,filename);
		//best[5][3];
		//keep reducing bins
		//int j = results.max_odds_i();
		//printf("\nThe best odds are: %e, which occur for nu of %e Hz and"
			   //" nudot of -%e Hz/s\n\n",
		//		results.odds[j], results.nu[j], results.nudot[j]);
		//printf("%d searches completed\n",counter);
		return 0;
	}
	catch(thrust::system_error &err)
	{
		std::cerr << "Error doing this: " << err.what() << std::endl;
		return 1;
	}
}


double t_odds(double *counts_h, int length,
		double nu, double nudot,
		int *mvals_h, int n_mvals)
{
	try
	{
		thrust::device_vector<double> counts_d(counts_h, counts_h+length);
		thrust::device_vector<unsigned char> t_binning(length*n_mvals,0);
		thrust::device_vector<int> mvals_d(mvals_h, mvals_h+n_mvals);
		double *counts_d_pointer = thrust::raw_pointer_cast(counts_d.data());
		unsigned char *t_binning_pointer = thrust::raw_pointer_cast(t_binning.data());
		int *mvals_d_pointer = thrust::raw_pointer_cast(mvals_d.data());
		t_bin_counts<<<40285,1024>>>(counts_d_pointer, length, t_binning_pointer, nu, nudot, 
									 mvals_d_pointer, n_mvals);
		//clear up space
		counts_d.clear();
		counts_d.shrink_to_fit();
		//iterate through segments of array
		thrust::device_vector<unsigned char>::iterator iter_start = t_binning.begin();
		thrust::device_vector<unsigned char>::iterator iter_end   = t_binning.begin();
		//sort parts of array
		for (int i = 0; i < n_mvals; i++)
		{
			thrust::advance(iter_end,length);
			thrust::sort(iter_start, iter_end);
			//thrust::sort(&t_binning[i*length],&t_binning[i*length + length]);
			thrust::advance(iter_start,length);
		}
		
		double odds = 0;
		iter_start = t_binning.begin();
		iter_end = t_binning.begin();
		for (int i = 0; i < n_mvals; i++)
		{
			thrust::advance(iter_end,length);
			thrust::device_vector<int> histogram(mvals_h[i],0);
			thrust::host_vector<unsigned char> histo_vals_h(mvals_h[i],0);
			for (unsigned char j = 0; j < mvals_h[i]; j++)
			{
				histo_vals_h[j] = j;
			}
			thrust::device_vector<unsigned char> histo_vals=histo_vals_h;
			thrust::reduce_by_key(iter_start, iter_end,	
					thrust::constant_iterator<int>(1),
					histo_vals.begin(),
					histogram.begin());
			thrust::advance(iter_start,length);
			//load these values back to the host, as has been binned
			thrust::host_vector<int> binned = histogram;
			double om1 = 0;
			for (int j = 0; j < binned.size(); j++)
			{
				om1+=logFacts[binned[j]];
			}
			om1  += logFacts[mvals_h[i]-1]-logFacts[length+mvals_h[i]-1]+((double)length)*log(mvals_h[i]);
			odds += exp(om1);
		}
		return odds;
	}
	catch(thrust::system_error &err)
	{
		std::cerr << "Error doing this: " << err.what() << std::endl;
		exit(-1);
	}

}

//function uploads static data to the GPU at start of MPI proc
void upload_data(double *counts_h, double *counts_d, int length,
				 int *mvals_h, int *mvals_d, int n_mvals)
{
	printf("1000 toa = %f\n",counts_h[1000]);
	//hipMalloc((void**)&counts_d,length*sizeof(double));		
	//hipMalloc((void**)&mvals_d,n_mvals*sizeof(double));
	//hipMemcpy(counts_d,counts_h,length*sizeof(double),
	// 		   hipMemcpyHostToDevice);
	//hipMemcpy(mvals_d,mvals_h,n_mvals*sizeof(double),
	//		   hipMemcpyHostToDevice);
	hipError_t error = hipGetLastError();
	if (error!=hipSuccess) {printf("Error! %s\n",hipGetErrorString(error));}
	else{printf("Static data uploaded!\n");}
}

void free_data(double *counts_d, int *mvals_d)
{
	//hipFree(mvals_d);
	//hipFree(counts_d);
}

double bins_to_odds(unsigned char *bins, int length, 
					int *mvals, int n_mvals)
{
	double odds = 0;
	for (int i = 0; i < n_mvals; i ++)
	{
		double om1 = 0;
		int m = mvals[i];	
		int n[m];
		for (int j = 0; j < m; j ++)
		{
			n[j] = 0;
		}
		for (int k = i*length; k < (i+1)*length; k++)
		{
			n[bins[k]]++;
		}
		/*
		#pragma omp parallel
		{
			int ni[m];
			for (int r = 0; r < m; r ++)
			{
				ni[r] = 0;
			}
			#pragma omp for
			for (int k = i*length; k < (i+1)*length; k++)
			{
				ni[bins[k]]++;
			}
			#pragma omp critical
			for (int q = 0; q < m; q++)
			{
				n[q] += ni[q];	
			}
		}
		*/
		for (int l = 0; l < m; l++)
		{
			//part of odds equation
			om1 += logFacts[n[l]];
		}
		om1 += logFacts[m-1]-logFacts[length+m-1]+((double)length)*log(m);
		odds += exp(om1);
	}
	return odds;
}

				

//Equation from gregory and loredo paper to calcluate odds ratio
//of m-binned stepwise model w.r.t. constant model
double log_m_odds_ratio(double *counts, int length, int m, 
					  double nu, double nudot,
					  double t_max)
{
	//create all the bins, init to zero counts
	unsigned int ng[m];

	//init to zero
	for (int j = 0; j < m; j++)
	{
		ng[j] = 0;
	}
	
	//split up into threads
	#pragma omp parallel default(shared)
	{
		//create temp bins for thread
		unsigned int n[m];
		for (int j = 0; j < m; j++)
		{
			n[j] = 0;
		}

		//variables used in binnings
		//gets position in nu
		//long double phi, d_phi;
		//double phi;
		//gets bin
		//int k;
		//bin the photons
		#pragma omp for 
		for (int i = 0; i < length; i++)
		{
			
			//d_phi = 0.5*counts[i]*nudot*counts[i];
			//get position in nu of photon
			//phi = fmod(counts[i]*nu+d_phi,1);
			//get corresponding bin	
			//k = (int)(fmod(counts[i]*(nu+0.5*counts[i]*nudot),1)*m);
			//one more count
			n[(int)(fmod(counts[i]*(nu+0.5*counts[i]*nudot),1)*m)]++;
			
		}

		//combine n values
		#pragma omp critical
		for (int j = 0; j < m; j++)
		{
			ng[j] += n[j];
		}
	}
	//odds to return
	double om1 = 0.0;
	//go through all bins
	for (int j = 0; j < m; j++)
	{
	
		//part of odds equation
		om1 += logFacts[ng[j]];
	}
	//final parts of odds equation
	om1 += logFacts[m-1]-logFacts[length+m-1]+((double)length)*log(m);
	return om1;
}

//Equation from gregory and loredo paper to calcluate total odds
//ratio
double log_odds_ratio(double *counts, int length, int *mvals, int n_mvals, 
					  double nu, double nudot, bool verbose)
{
	//normalize the counts with item 0 at t=0.0s
	normalize_counts(counts, length);
	//the following assumes the counts are ordered
	double t_max = counts[length-1];
	//The total odds ratio
	double odds = 0.0;
	//go through all possible m values
	for (int i = 0; i <= n_mvals; i++)
	{
		if (verbose)
			printf("Testing %d-binned model\n",i);
		//Add the next om1 value to the total odds ratio.
		//We also have to remove the log

		odds += exp(log_m_odds_ratio(counts,length,mvals[i],nu,
									 nudot,t_max));
	}
	return odds;
}

//Gets the average time between counts
double avg_interval(double *counts, int length)
{
	double total_time;
	total_time = counts[length-1] - counts[0];
	return total_time/length;
}

//Gets the minimum time between counts
double min_interval(double *counts, int length)
{
	double smallest;
	//start with first interval
	smallest = counts[1] - counts[0];
	//go through the rest
	for (int i = 2; i < length; i ++)
	{
		double tmp = counts[i] - counts[i-1];
		//if interval smaller, assume it is now
		//the smallest
		if (tmp < smallest)
		{
			smallest = tmp;
		}
	}
	return smallest;
}

